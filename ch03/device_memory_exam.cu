#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


void checkDeviceMemory(void)
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void)
{
    int* dDataPtr;
    hipError_t errorCode;

    checkDeviceMemory();
    errorcode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("hipMemset - %s\n", hipGetErrorName(errorCode));

    errorCode = hipFree(dDataPtr);
    printf("hipFree - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();
}