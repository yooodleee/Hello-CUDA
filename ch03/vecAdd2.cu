#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>


// The size of the vec
#define NUM_DATA 1024


// Simple vec sum kernel
__global__ void vecAdd(int* _a, int* _b, int* _c)
{
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    // Set timer
    DS_timer timer(5);
    timer.setTimerName(0, (char*)"CUDA Total");
    timer.setTimerName(1, (char*)"Computation(Kernel)");
    timer.setTimerName(2, (char*)"Data Trans. : Host -> Device");
    timer.setTimerName(3, (char*)"Data Trans. : Device -> Host");
    timer.setTimerName(4, (char*)"VecAdd on Host");
    timer.initTimers();

    int* a, * b, * c, * hc;     // Vecs on the host
    int* da, * db, * dc;        // Vecs on the device

    int memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // Memory allocation on the host-side
    a = new int[NUM_DATA]; memset(a, 0, memSize);
    b = new int[NUM_DATA]; memset(b, 0, memSize);
    c = new int[NUM_DATA]; memset(c, 0, memSize);
    hc = new int[NUM_DATA]; memset(hc, 0, memSize);

    // Data generation
    for (int i = 0; i < NUM_DATA; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Vec sum on host (for performance comparison)
    timer.onTimer(4);
    for (int i = 0; i < NUM_DATA; i++)
        hc[i] = a[i] + b[i];
    timer.offTimer(4);

    // Memory allocation on the device-side
    hipMalloc(&da, memSize); hipMemset(da, 0, memSize);
    hipMalloc(&db, memSize); hipMemset(db, 0, memSize);
    hipMalloc(&dc, memSize); hipMemset(dc, 0, memSize);

    timer.onTimer(0);

    // Data copy : Host -> Device
    timer.onTimer(2);
    hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);
    timer.offTimer(2);

    // Kernel call
    timer.onTimer(1);
    vecAdd <<< 1, NUM_DATA >>> (da, db, dc);
    hipDeviceSynchronize();
    timer.offTimer(1);

    // Copy results: Device -> Host
    timer.onTimer(3);
    hipMemcpy(c, dc, memSize, hipMemcpyHostToDevice);
    timer.onTimer(3);

    timer.offTimer(0);

    // Release device memory
    hipFree(da); hipFree(db); hipFree(dc);

    timer.printTimer();
    
    // Check results
    bool result = true;
    for (int i = 0; i < NUM_DATA; i++)
    {
        if (hc[i] != c[i])
        {
            printf("[%d] The result is not mathced! (%d, %d)\n", i, hc[i], c[i]);
            result = false;
        }
    }

    if (result)
        printf("GPU works well!\n");

    // Release host memory
    delete[] a; delete[] b; delete[] c;

    return 0;
}