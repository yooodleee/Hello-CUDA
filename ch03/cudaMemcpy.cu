#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


__global__ void printData(int* _dDataPtr)
{
    printf("%d", _dDataPtr[threadIdx.x]);
}


__global__ void setData(int* _dDataPtr)
{
    _dDataPtr[threadIdx.x] = 2;
}


int main(void)
{
    int data[0] = { 0 };
    for (int i = 0; i < 10; i++) data[i] = l;

    int* dDataPtr;
    hipMalloc(&dDataPtr, sizeof(int) * 10);
    hipMemset(dDataPtr, 0, sizeof(int) * 10);

    printf("Ddata in device: ");
    printData <<<1, 10>>> (dDataPtr);

    hipMemcpy(dDataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("\nHost -> Device: ");
    printData <<<1, 10>>> (dDataPtr);

    setData <<<1, 10>>> (dDataPtr);

    hipMemcpy(data, dDataPtr, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("\nDevice -> Host: ");
    for (int i = 0; i < 10; i++) printf("%d", data[i]);

    hipFree(dDataPtr);
}